#include "hip/hip_runtime.h"
﻿
//generic c/c++ libs
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
//opengl / glm
#include "include/glad/glad.h"
#include "include/GLFW/glfw3.h"
#include "include/glm/glm.hpp"
#include "include/glm/gtc/matrix_transform.hpp"
#include "include/glm/gtc/type_ptr.hpp"
//cuda
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_fp16.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/gather.h>
#include <thrust/iterator/zip_iterator.h>
#include <cuda_gl_interop.h>
//code-specific libraries
#include "camera/transformations.h"
#include "camera/camera.h"
#include "camera/Features.h"
#include "settings/settings.h"
#include "shaders/Shader.h"
#include "shaders/Buffer.h"
#include "settings/controls.h"
#include "lbm/meshSetup.h"
#include "inputGeometry/geometry.h"
#include "render/renderSetup.h"
#include "render/visualize.h"
#include "lbm/meshSetup.h"


int main()
{
    // glfw window creation
    window = createWindow();
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glEnable(GL_BLEND);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_HIDDEN);
    projection = glm::perspective(glm::radians(45.0f), 800.0f / 600.0f, 0.1f, 400.0f);
    camera = Features::GetCamera();
    //shaders
    Shader vertex("shaders/vertex.glsl", "shaders/fragment.glsl");
    Shader vertex2("shaders/vertex2.glsl", "shaders/fragment.glsl");
    //geometry
    VertexList v = read_verticesV(path);
    Buffer buff(v.vertices, nullptr, 4 * v.size, 1);
    GLuint vao, vbo;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, NX*NY*NZ*16, nullptr, GL_DYNAMIC_DRAW);
    // Specify the vertex attribute layout
    // current vertex array stores 3 vec3s for vertices and 1 float for color
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
    glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(3 * 4));
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);
    hipGraphicsResource* cuda_vbo_resource;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard);
    size_t num_bytes;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, cuda_vbo_resource);
    //initialize marching cubes for rendering
    mcr::begin();
    __half* sc = (__half*)calloc((NX + 1) * (NY + 1) * (NZ + 1), sizeof(__half));
    hipMalloc(&sc_field, sizeof(__half) * (NX + 1) * (NY + 1) * (NZ + 1));
    hipMemcpy(sc_field, sc, sizeof(__half) * (NX + 1) * (NY + 1) * (NZ + 1), hipMemcpyHostToDevice);
   // mcr::marchCubes(sc_field, dptr);
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
    free(sc);
    //initialize LBM cfd 
    lbm::begin(v.vertices, v.size);  
    while (!glfwWindowShouldClose(window)){
        //generic camera calculations for translation/rotation
        glm::mat4 inverse = glm::mat4(1);
        inverse[0] = camera.camera_basis.basis[0];
        inverse[1] = camera.camera_basis.basis[1];
        inverse[2] = camera.camera_basis.basis[2];
        inverse[3] = glm::vec4(0, 0, 0, 1);
        inverse = glm::transpose(inverse);
        if (launched) {    
            hipGraphicsMapResources(1, &cuda_vbo_resource, 0); // Map the VBO resource
            hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, cuda_vbo_resource);
            compute();//perform LBM calculations
            mcr::marchCubes(sc_field, dptr); //render the scalar field
            hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);          
            processInput(window);
            glClearColor(0.0f, 0.0f, 0.1f, 1.0f);
            glClear(GL_COLOR_BUFFER_BIT);
            glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
            glUseProgram(vertex.ID);
            glBindVertexArray(buff.VAO);
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "projection"), 1, GL_FALSE, &projection[0][0]);
            glUniform3fv(glGetUniformLocation(vertex2.ID, "camera_global"), 1, glm::value_ptr(glm::vec3(camera.camera_basis.basis[3].x, camera.camera_basis.basis[3].y, camera.camera_basis.basis[3].z)));
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "view_direction"), 1, GL_FALSE, glm::value_ptr(inverse));
            glDrawArrays(GL_TRIANGLES, 0, 4 * v.size);   //render the mesh        
            glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
            glUseProgram(vertex2.ID);
            glBindVertexArray(vao);
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "projection"), 1, GL_FALSE, &projection[0][0]);
            glUniform3fv(glGetUniformLocation(vertex2.ID, "camera_global"), 1, glm::value_ptr(glm::vec3(camera.camera_basis.basis[3].x, camera.camera_basis.basis[3].y, camera.camera_basis.basis[3].z)));
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "view_direction"), 1, GL_FALSE, glm::value_ptr(inverse));
            glDrawArrays(GL_TRIANGLES, 0, (*ntriangle_count)); //render the field
            //read to file if desired
            if (COMPUTATION_MODE) {
                glReadPixels(0, 0, SCR_WIDTH, SCR_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, image);
                fwrite(image, 1, SCR_WIDTH * SCR_HEIGHT * 4, ffmpeg);
            }                                   
        }
        else {
            processInput(window);        
            glClearColor(0.0f, 0.0f, 0.1f, 1.0f);
            glClear(GL_COLOR_BUFFER_BIT);
            glUseProgram(vertex.ID);
            glBindVertexArray(buff.VAO);
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "projection"), 1, GL_FALSE, &projection[0][0]);
            glUniform3fv(glGetUniformLocation(vertex2.ID, "camera_global"), 1, glm::value_ptr(glm::vec3(camera.camera_basis.basis[3].x, camera.camera_basis.basis[3].y, camera.camera_basis.basis[3].z)));
            glUniformMatrix4fv(glGetUniformLocation(vertex2.ID, "view_direction"), 1, GL_FALSE, glm::value_ptr(inverse));
            glDrawArrays(GL_TRIANGLES, 0, 4 * v.size);
        }      
        glfwPollEvents();
        HandleSettings(window, camera);
        PollControls(camera, window);
        glfwSwapBuffers(window);          
    }
    _pclose(ffmpeg);
    lbm::__terminate__();
    mcr::__terminate__();
    return 0;
}
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

